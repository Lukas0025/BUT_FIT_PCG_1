#include "hip/hip_runtime.h"
/**
 * @file      nbody.cu
 *
 * @author    Name Surname \n
 *            Faculty of Information Technology \n
 *            Brno University of Technology \n
 *            xpleva07@vutbr.cz
 *
 * @brief     PCG Assignment 1
 *
 * @version   2023
 *
 * @date      04 October   2023, 09:00 (created) \n
 */

#include <>

#include "nbody.cuh"

#define FLOAT_MIN 1.17549e-38
#define FULL_MASK 0xffffffff

/* Constants */
constexpr float G                  = 6.67384e-11f;
constexpr float COLLISION_DISTANCE = 0.01f;

/**
 * CUDA kernel to calculate new particles velocity and position
 * @param pIn  - particles in
 * @param pOut - particles out
 * @param N    - Number of particles
 * @param dt   - Size of the time step
 */
__global__ void calculateVelocity(Particles pIn, Particles pOut, const unsigned N, float dt)
{
 // determinate ID of thread and total number of threads
  const unsigned ix     = threadIdx.x + blockIdx.x * blockDim.x;
  const unsigned stride = gridDim.x * blockDim.x;

  //shared particles
  extern __shared__ float s[];

  // for simple indexing
  float* const inPosX    = pIn.posX;
  float* const inPosY    = pIn.posY;
  float* const inPosZ    = pIn.posZ;
  float* const inVelZ    = pIn.velZ;

  // in faster shared memory
  float* const inWeight  = s;
  float* const inVelX    = &(inWeight[N]);
  float* const inVelY    = &(inVelX[N]);

  float* const outPosX   = pOut.posX;
  float* const outPosY   = pOut.posY;
  float* const outPosZ   = pOut.posZ;
  float* const outVelX   = pOut.velX;
  float* const outVelY   = pOut.velY;
  float* const outVelZ   = pOut.velZ;
  float* const outWeight = pOut.weight;

  // load particles to shared memory
  for (unsigned i = threadIdx.x; i < N; i += blockDim.x) {
    inVelX[i]    = pIn.velX[i];
    inVelY[i]    = pIn.velY[i];
    inWeight[i]  = pIn.weight[i];
  }

  // wait until load particles to shared memory
  __syncthreads();
  
  // iterate over all object for one threat
  for (unsigned i = ix; i < N; i += stride) {
    float newVelX = 0;
    float newVelY = 0;
    float newVelZ = 0;

    float colisionVelX = 0;
    float colisionVelY = 0;
    float colisionVelZ = 0;

    const float posX   = inPosX[i];
    const float posY   = inPosY[i];
    const float posZ   = inPosZ[i];
    const float velX   = inVelX[i];
    const float velY   = inVelY[i];
    const float velZ   = inVelZ[i];
    const float weight = inWeight[i];

    // iterate over all objects
    for (unsigned j = 0u; j < N; ++j) {
      const float otherPosX   = inPosX[j];
      const float otherPosY   = inPosY[j];
      const float otherPosZ   = inPosZ[j];
      const float otherVelX   = inVelX[j];
      const float otherVelY   = inVelY[j];
      const float otherVelZ   = inVelZ[j];
      const float otherWeight = inWeight[j];

      const float dx = otherPosX - posX;
      const float dy = otherPosY - posY;
      const float dz = otherPosZ - posZ;

      const float r2 = dx * dx + dy * dy + dz * dz;
      const float r = sqrt(r2) + FLOAT_MIN; // to awoid zero div

      const float f = G * weight * otherWeight / r2 + FLOAT_MIN; // to awoid zero div

      // calculate new velocity
      newVelX += (r > COLLISION_DISTANCE) ? dx / r * f : 0;
      newVelY += (r > COLLISION_DISTANCE) ? dy / r * f : 0;
      newVelZ += (r > COLLISION_DISTANCE) ? dz / r * f : 0;
      
      colisionVelX += (r > 0.f && r < COLLISION_DISTANCE) ? 
        (((weight * velX - otherWeight * velX + 2.f * otherWeight * otherVelX) / (weight + otherWeight)) - velX) : 0;
      colisionVelY += (r > 0.f && r < COLLISION_DISTANCE) ?
        (((weight * velY - otherWeight * velY + 2.f * otherWeight * otherVelY) / (weight + otherWeight)) - velY) : 0;
      colisionVelZ += (r > 0.f && r < COLLISION_DISTANCE) ? 
        (((weight * velZ - otherWeight * velZ + 2.f * otherWeight * otherVelZ) / (weight + otherWeight)) - velZ) : 0;
    }

    newVelX *= dt / weight;
    newVelY *= dt / weight;
    newVelZ *= dt / weight;

    //colisition update speed

    newVelX += colisionVelX;
    newVelY += colisionVelY;
    newVelZ += colisionVelZ;

    //update position

    outVelX[i]   = velX + newVelX;
    outVelY[i]   = velY + newVelY;
    outVelZ[i]   = velZ + newVelZ;
    outWeight[i] = weight;

    outPosX[i]   = posX + outVelX[i] * dt;
    outPosY[i]   = posY + outVelY[i] * dt;
    outPosZ[i]   = posZ + outVelZ[i] * dt;
  }
}// end of calculate_gravitation_velocity
//----------------------------------------------------------------------------------------------------------------------

/**
 * Kernel to calculate particles center of mass
 * @param p    - particles
 * @param N    - Number of particles
 */
__device__ static inline void centerOfMassReduction(float4* a, const float4* b)
{
  float4 d = {b->x - a->x,
              b->y - a->y,
              b->z - a->z,
              (a->w + b->w) > 0.f ? (b->w / (a->w + b->w)) : 0.f};

  a->x += d.x * d.w;
  a->y += d.y * d.w;
  a->z += d.z * d.w;
  a->w += b->w;
}

/**
 * CUDA kernel to calculate particles center of mass
 * @param p    - particles
 * @param com  - pointer to a center of mass
 * @param lock - pointer to a user-implemented lock
 * @param N    - Number of particles
 */
__global__ void centerOfMass(Particles p, float4* com, int* lock, const unsigned N)
{
  /********************************************************************************************************************/
  /*           TODO: CUDA kernel to calculate particles center of mass, see reference CPU implementation,             */
  /*                                 use CUDA predefined warpSize variable                                            */
  /********************************************************************************************************************/
  
  // determinate ID of thread and total number of threads
  const unsigned ix     = threadIdx.x + blockIdx.x * blockDim.x;
  const unsigned stride = gridDim.x * blockDim.x;

  extern __shared__ float4 shm[];

  // for simple indexing
  float* const pPosX   = p.posX;
  float* const pPosY   = p.posY;
  float* const pPosZ   = p.posZ;
  float* const pWeight = p.weight;

  float4 local_com = {0, 0, 0, 0};

  // iterate over all object for one threat
  for (unsigned i = ix; i < N; i += stride) {
    const float4 particle = {pPosX[i], pPosY[i], pPosZ[i], pWeight[i]};

    centerOfMassReduction(&local_com, &particle);
  }

  __syncthreads(); // wait until all done

  // Warp-synchronous reduction.
  # pragma unroll
  //for (unsigned stride = warpSize / 2; stride > 0; stride = stride / 2) {
  for (unsigned stride = warpSize >> 1; stride > 0; stride = stride >> 1) {
    float4 other_com = {
      __shfl_down_sync(FULL_MASK, local_com.x, stride),
      __shfl_down_sync(FULL_MASK, local_com.y, stride),
      __shfl_down_sync(FULL_MASK, local_com.z, stride),
      __shfl_down_sync(FULL_MASK, local_com.w, stride)
    };

    centerOfMassReduction(&local_com, &other_com);
    
    __syncthreads(); // wait until all done
  }

  if ((threadIdx.x & 31) != 0) return; // keep only first threatds in warps

  // reduction on shared memory
  unsigned warpId     = threadIdx.x / warpSize; // << 5
  unsigned warpsCount = blockDim.x  / warpSize + 0.5;
  shm[warpId]         = local_com;

  for (unsigned stride = warpsCount >> 1; stride > 0; stride = stride >> 1) {
    if (warpId < stride) {
      centerOfMassReduction(&(shm[warpId]), &(shm[warpId + stride]));
    }

    __syncthreads();
  }

  if (warpId == 0) { // otput of reduction position
    while (atomicCAS(lock,0,1) != 0); //lock
    
    centerOfMassReduction(com, shm);

    atomicExch(lock, 0); // unlock

  }

}// end of centerOfMass
//----------------------------------------------------------------------------------------------------------------------

/**
 * CPU implementation of the Center of Mass calculation
 * @param particles - All particles in the system
 * @param N         - Number of particles
 */
__host__ float4 centerOfMassRef(MemDesc& memDesc)
{
  float4 com{};

  for (std::size_t i{}; i < memDesc.getDataSize(); i++)
  {
    const float3 pos = {memDesc.getPosX(i), memDesc.getPosY(i), memDesc.getPosZ(i)};
    const float  w   = memDesc.getWeight(i);

    // Calculate the vector on the line connecting current body and most recent position of center-of-mass
    // Calculate weight ratio only if at least one particle isn't massless
    const float4 d = {pos.x - com.x,
                      pos.y - com.y,
                      pos.z - com.z,
                      ((memDesc.getWeight(i) + com.w) > 0.0f)
                        ? ( memDesc.getWeight(i) / (memDesc.getWeight(i) + com.w))
                        : 0.0f};

    // Update position and weight of the center-of-mass according to the weight ration and vector
    com.x += d.x * d.w;
    com.y += d.y * d.w;
    com.z += d.z * d.w;
    com.w += w;
  }

  return com;
}// enf of centerOfMassRef
//----------------------------------------------------------------------------------------------------------------------
