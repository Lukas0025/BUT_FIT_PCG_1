#include "hip/hip_runtime.h"
/**
 * @file      nbody.cu
 *
 * @author    Name Surname \n
 *            Faculty of Information Technology \n
 *            Brno University of Technology \n
 *            xlogin00@fit.vutbr.cz
 *
 * @brief     PCG Assignment 1
 *
 * @version   2023
 *
 * @date      04 October   2023, 09:00 (created) \n
 */

#include <>

#include "nbody.cuh"

/* Constants */
constexpr float G                  = 6.67384e-11f;
constexpr float COLLISION_DISTANCE = 0.01f;

/**
 * CUDA kernel to calculate gravitation velocity
 * @param p      - particles
 * @param tmpVel - temp array for velocities
 * @param N      - Number of particles
 * @param dt     - Size of the time step
 */
__global__ void calculateGravitationVelocity(Particles p, Velocities tmpVel, const unsigned N, float dt)
{
  float* const pPosX   = p.posX;
  float* const pPosY   = p.posY;
  float* const pPosZ   = p.posZ;
  float* const pVelX   = p.velX;
  float* const pVelY   = p.velY;
  float* const pVelZ   = p.velZ;
  float* const pWeight = p.weight;

  float* const tmpVelX = tmpVel.x;
  float* const tmpVelY = tmpVel.y;
  float* const tmpVelZ = tmpVel.z;

  for (unsigned i = 0u; i < N; ++i) { // here determinate particle by ID
    float newVelX{};
    float newVelY{};
    float newVelZ{};

    const float posX   = pPosX[i];
    const float posY   = pPosY[i];
    const float posZ   = pPosZ[i];
    const float weight = pWeight[i];

    for (unsigned j = 0u; j < N; ++j) {
      const float otherPosX   = pPosX[j];
      const float otherPosY   = pPosY[j];
      const float otherPosZ   = pPosZ[j];
      const float otherWeight = pWeight[j];

      const float dx = otherPosX - posX;
      const float dy = otherPosY - posY;
      const float dz = otherPosZ - posZ;

      const float r2 = dx * dx + dy * dy + dz * dz;
      const float r = std::sqrt(r2) + std::numeric_limits<float>::min();

      const float f = G * weight * otherWeight / r2 + std::numeric_limits<float>::min();

      newVelX += (r > COLLISION_DISTANCE) ? dx / r * f : 0.f;
      newVelY += (r > COLLISION_DISTANCE) ? dy / r * f : 0.f;
      newVelZ += (r > COLLISION_DISTANCE) ? dz / r * f : 0.f;
    }

    newVelX *= dt / weight;
    newVelY *= dt / weight;
    newVelZ *= dt / weight;

    tmpVelX[i] = newVelX;
    tmpVelY[i] = newVelY;
    tmpVelZ[i] = newVelZ;
  }  
}// end of calculate_gravitation_velocity
//----------------------------------------------------------------------------------------------------------------------

/**
 * CUDA kernel to calculate collision velocity
 * @param p      - particles
 * @param tmpVel - temp array for velocities
 * @param N      - Number of particles
 * @param dt     - Size of the time step
 */
__global__ void calculateCollisionVelocity(Particles p, Velocities tmpVel, const unsigned N, float dt) {
  
  float* const pPosX   = p.posX;
  float* const pPosY   = p.posY;
  float* const pPosZ   = p.posZ;
  float* const pVelX   = p.velX;
  float* const pVelY   = p.velY;
  float* const pVelZ   = p.velZ;
  float* const pWeight = p.weight;

  float* const tmpVelX = tmpVel.x;
  float* const tmpVelY = tmpVel.y;
  float* const tmpVelZ = tmpVel.z;

  for (unsigned i = 0u; i < N; ++i) {
    float newVelX{};
    float newVelY{};
    float newVelZ{};

    const float posX   = pPosX[i];
    const float posY   = pPosY[i];
    const float posZ   = pPosZ[i];
    const float velX   = pVelX[i];
    const float velY   = pVelY[i];
    const float velZ   = pVelZ[i];
    const float weight = pWeight[i];

    for (unsigned j = 0u; j < N; ++j) {
      const float otherPosX   = pPosX[j];
      const float otherPosY   = pPosY[j];
      const float otherPosZ   = pPosZ[j];
      const float otherVelX   = pVelX[j];
      const float otherVelY   = pVelY[j];
      const float otherVelZ   = pVelZ[j];
      const float otherWeight = pWeight[j];

      const float dx = otherPosX - posX;
      const float dy = otherPosY - posY;
      const float dz = otherPosZ - posZ;

      const float r2 = dx * dx + dy * dy + dz * dz;
      const float r = std::sqrt(r2);

      newVelX += (r > 0.f && r < COLLISION_DISTANCE)
                 ? (((weight * velX - otherWeight * velX + 2.f * otherWeight * otherVelX) / (weight + otherWeight)) - velX)
                 : 0.f;
      newVelY += (r > 0.f && r < COLLISION_DISTANCE)
                 ? (((weight * velY - otherWeight * velY + 2.f * otherWeight * otherVelY) / (weight + otherWeight)) - velY)
                 : 0.f;
      newVelZ += (r > 0.f && r < COLLISION_DISTANCE)
                 ? (((weight * velZ - otherWeight * velZ + 2.f * otherWeight * otherVelZ) / (weight + otherWeight)) - velZ)
                 : 0.f;
    }

    tmpVelX[i] += newVelX;
    tmpVelY[i] += newVelY;
    tmpVelZ[i] += newVelZ;
  }
}// end of calculate_collision_velocity
//----------------------------------------------------------------------------------------------------------------------

/**
 * CUDA kernel to update particles
 * @param p      - particles
 * @param tmpVel - temp array for velocities
 * @param N      - Number of particles
 * @param dt     - Size of the time step
 */
__global__ void updateParticles(Particles p, Velocities tmpVel, const unsigned N, float dt) {
  
  float* const pPosX   = p.posX;
  float* const pPosY   = p.posY;
  float* const pPosZ   = p.posZ;
  float* const pVelX   = p.velX;
  float* const pVelY   = p.velY;
  float* const pVelZ   = p.velZ;
  float* const pWeight = p.weight;

  float* const tmpVelX = tmpVel.x;
  float* const tmpVelY = tmpVel.y;
  float* const tmpVelZ = tmpVel.z;

  for (unsigned i = 0u; i < N; ++i) {

    float posX = pPosX[i];
    float posY = pPosY[i];
    float posZ = pPosZ[i];

    float velX = pVelX[i];
    float velY = pVelY[i];
    float velZ = pVelZ[i];

    const float newVelX = tmpVelX[i];
    const float newVelY = tmpVelY[i];
    const float newVelZ = tmpVelZ[i];

    velX += newVelX;
    velY += newVelY;
    velZ += newVelZ;

    posX += velX * dt;
    posY += velY * dt;
    posZ += velZ * dt;

    pPosX[i] = posX;
    pPosY[i] = posY;
    pPosZ[i] = posZ;

    pVelX[i] = velX;
    pVelY[i] = velY;
    pVelZ[i] = velZ;
  }
}// end of update_particle
//----------------------------------------------------------------------------------------------------------------------

/**
 * CUDA kernel to calculate particles center of mass
 * @param p    - particles
 * @param com  - pointer to a center of mass
 * @param lock - pointer to a user-implemented lock
 * @param N    - Number of particles
 */
__global__ void centerOfMass(Particles p, float4* com, int* lock, const unsigned N) {
  float4 d = {b.x - a.x,
              b.y - a.y,
              b.z - a.z,
              (a.w + b.w) > 0.f ? (b.w / (a.w + b.w)) : 0.f};

  a.x += d.x * d.w;
  a.y += d.y * d.w;
  a.z += d.z * d.w;
  a.w += b.w;
}// end of centerOfMass
//----------------------------------------------------------------------------------------------------------------------

/**
 * CPU implementation of the Center of Mass calculation
 * @param particles - All particles in the system
 * @param N         - Number of particles
 */
__host__ float4 centerOfMassRef(MemDesc& memDesc)
{
  float4 com{};

  for (std::size_t i{}; i < memDesc.getDataSize(); i++)
  {
    const float3 pos = {memDesc.getPosX(i), memDesc.getPosY(i), memDesc.getPosZ(i)};
    const float  w   = memDesc.getWeight(i);

    // Calculate the vector on the line connecting current body and most recent position of center-of-mass
    // Calculate weight ratio only if at least one particle isn't massless
    const float4 d = {pos.x - com.x,
                      pos.y - com.y,
                      pos.z - com.z,
                      ((memDesc.getWeight(i) + com.w) > 0.0f)
                        ? ( memDesc.getWeight(i) / (memDesc.getWeight(i) + com.w))
                        : 0.0f};

    // Update position and weight of the center-of-mass according to the weight ration and vector
    com.x += d.x * d.w;
    com.y += d.y * d.w;
    com.z += d.z * d.w;
    com.w += w;
  }

  return com;
}// enf of centerOfMassRef
//----------------------------------------------------------------------------------------------------------------------
