#include "hip/hip_runtime.h"
/**
 * @file      main.cu
 *
 * @author    Name Surname \n
 *            Faculty of Information Technology \n
 *            Brno University of Technology \n
 *            xlogin00@fit.vutbr.cz
 *
 * @brief     PCG Assignment 1
 *
 * @version   2023
 *
 * @date      04 October   2023, 09:00 (created) \n
 */

#include <cmath>
#include <cstdio>
#include <chrono>
#include <string>

#include "nbody.cuh"
#include "h5Helper.h"

/**
 * @brief CUDA error checking macro
 * @param call CUDA API call
 */
#define CUDA_CALL(call) \
  do { \
    const hipError_t _error = (call); \
    if (_error != hipSuccess) \
    { \
      std::fprintf(stderr, "CUDA error (%s:%d): %s\n", __FILE__, __LINE__, hipGetErrorString(_error)); \
      std::exit(EXIT_FAILURE); \
    } \
  } while(0)

/**
 * Main rotine
 * @param argc
 * @param argv
 * @return
 */
int main(int argc, char **argv)
{
  if (argc != 10)
  {
    std::printf("Usage: nbody <N> <dt> <steps> <threads/block> <write intesity> <reduction threads> <reduction threads/block> <input> <output>\n");
    std::exit(1);
  }

  // Number of particles
  const unsigned N                   = static_cast<unsigned>(std::stoul(argv[1]));
  // Length of time step
  const float    dt                  = std::stof(argv[2]);
  // Number of steps
  const unsigned steps               = static_cast<unsigned>(std::stoul(argv[3]));
  // Number of thread blocks
  const unsigned simBlockDim         = static_cast<unsigned>(std::stoul(argv[4]));
  // Write frequency
  const unsigned writeFreq           = static_cast<unsigned>(std::stoul(argv[5]));
  // number of reduction threads
  const unsigned redTotalThreadCount = static_cast<unsigned>(std::stoul(argv[6]));
  // Number of reduction threads/blocks
  const unsigned redBlockDim         = static_cast<unsigned>(std::stoul(argv[7]));

  // Size of the simulation CUDA grid - number of blocks
  const unsigned simGridDim = (N + simBlockDim - 1) / simBlockDim;
  // Size of the reduction CUDA grid - number of blocks
  const unsigned redGridDim = (redTotalThreadCount + redBlockDim - 1) / redBlockDim;

  // Log benchmark setup
  std::printf("       NBODY GPU simulation\n"
              "N:                       %u\n"
              "dt:                      %f\n"
              "steps:                   %u\n"
              "threads/block:           %u\n"
              "blocks/grid:             %u\n"
              "reduction threads/block: %u\n"
              "reduction blocks/grid:   %u\n",
              N, dt, steps, simBlockDim, simGridDim, redBlockDim, redGridDim);

  const std::size_t recordsCount = (writeFreq > 0) ? (steps + writeFreq - 1) / writeFreq : 0;

  Particles hParticles{};

  /********************************************************************************************************************/
  /*                                    CPU side memory allocation (pinned)                                           */
  /********************************************************************************************************************/

  // host particles
  hParticles.posX   = static_cast<float*>(operator new[](N * sizeof(float)));
  hParticles.posY   = static_cast<float*>(operator new[](N * sizeof(float)));
  hParticles.posZ   = static_cast<float*>(operator new[](N * sizeof(float)));
  hParticles.velX   = static_cast<float*>(operator new[](N * sizeof(float)));
  hParticles.velY   = static_cast<float*>(operator new[](N * sizeof(float)));
  hParticles.velZ   = static_cast<float*>(operator new[](N * sizeof(float)));
  hParticles.weight = static_cast<float*>(operator new[](N * sizeof(float)));

  float4 hFinalCom;

  /********************************************************************************************************************/
  /*                              TODO: Fill memory descriptor layout                                                 */
  /********************************************************************************************************************/
  /*
   * Caution! Create only after CPU side allocation
   * parameters:
   *                            Stride of two            Offset of the first
   *       Data pointer       consecutive elements        element in FLOATS,
   *                          in FLOATS, not bytes            not bytes
  */
  MemDesc md(hParticles.posX,           1,                         0,
             hParticles.posY,           1,                         0,
             hParticles.posZ,           1,                         0,
             hParticles.velX,           1,                         0,
             hParticles.velY,           1,                         0,
             hParticles.velZ,           1,                         0,
             hParticles.weight,         1,                         0,
             N,
             recordsCount);

  // Initialisation of helper class and loading of input data
  H5Helper h5Helper(argv[8], argv[9], md);

  try
  {
    h5Helper.init();
    h5Helper.readParticleData();
  }
  catch (const std::exception& e)
  {
    std::fprintf(stderr, "Error: %s\n", e.what());
    return EXIT_FAILURE;
  }

  Particles  dParticles{};
  Velocities dTmpVelocities{};
  float4    *dFinalCom;
  int       *dLock;

  /********************************************************************************************************************/
  /*                                           GPU side memory allocation                                             */
  /********************************************************************************************************************/

  // dParticles
  CUDA_CALL(hipMalloc(&(dParticles.posX),  N * sizeof(float)));
  CUDA_CALL(hipMalloc(&(dParticles.posY),  N * sizeof(float)));
  CUDA_CALL(hipMalloc(&(dParticles.posZ),  N * sizeof(float)));
  CUDA_CALL(hipMalloc(&(dParticles.velX),  N * sizeof(float)));
  CUDA_CALL(hipMalloc(&(dParticles.velY),  N * sizeof(float)));
  CUDA_CALL(hipMalloc(&(dParticles.velZ),  N * sizeof(float)));
  CUDA_CALL(hipMalloc(&(dParticles.weight), N * sizeof(float)));

  // dTmpVelocities
  CUDA_CALL(hipMalloc(&(dTmpVelocities.x), N * sizeof(float)));
  CUDA_CALL(hipMalloc(&(dTmpVelocities.y), N * sizeof(float)));
  CUDA_CALL(hipMalloc(&(dTmpVelocities.z), N * sizeof(float)));

  CUDA_CALL(hipMalloc(&dFinalCom, sizeof(float4)));
  CUDA_CALL(hipMalloc(&dLock, sizeof(int)));

  /********************************************************************************************************************/
  /*                                           Memory transfer CPU -> GPU                                             */
  /********************************************************************************************************************/

  // Particles
  CUDA_CALL(hipMemcpy(dParticles.posX,   hParticles.posX,   N * sizeof(float), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(dParticles.posY,   hParticles.posY,   N * sizeof(float), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(dParticles.posZ,   hParticles.posZ,   N * sizeof(float), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(dParticles.velX,   hParticles.velX,   N * sizeof(float), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(dParticles.velY,   hParticles.velY,   N * sizeof(float), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(dParticles.velZ,   hParticles.velZ,   N * sizeof(float), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(dParticles.weight, hParticles.weight, N * sizeof(float), hipMemcpyHostToDevice));

  CUDA_CALL(hipMemset(dFinalCom, 0, sizeof(float4)));
  CUDA_CALL(hipMemset(dLock, 0, sizeof(int)));
  
  // wait until done
  CUDA_CALL(hipDeviceSynchronize());

  // Lambda for checking if we should write current step to the file
  auto shouldWrite = [writeFreq](unsigned s) -> bool
  {
    return writeFreq > 0u && (s % writeFreq == 0u);
  };

  // Lamda for getting record number
  auto getRecordNum = [writeFreq](unsigned s) -> unsigned
  {
    return s / writeFreq;
  };

  // Start measurement
  const auto start = std::chrono::steady_clock::now();

  for (unsigned s = 0u; s < steps; ++s) {
    if (shouldWrite(s)) {
      const auto recordNum = getRecordNum(s);

      centerOfMass <<< 32, 64 >>> (dParticles, dFinalCom, dLock, N);

      CUDA_CALL(hipMemcpy(&hFinalCom, dFinalCom, sizeof(float4), hipMemcpyDeviceToHost));
      CUDA_CALL(hipMemset(dFinalCom, 0, sizeof(float4)));

      h5Helper.writeParticleData(recordNum);
      h5Helper.writeCom(hFinalCom, recordNum);
    }

    calculateGravitationVelocity <<< 32, 64 >>> (dParticles, dTmpVelocities, N, dt);
      CUDA_CALL(hipDeviceSynchronize());
    calculateCollisionVelocity   <<< 32, 64 >>> (dParticles, dTmpVelocities, N, dt);
      CUDA_CALL(hipDeviceSynchronize());
    updateParticles              <<< 32, 64 >>> (dParticles, dTmpVelocities, N, dt);
  }

  // Wait for all CUDA kernels to finish
  CUDA_CALL(hipDeviceSynchronize());

  // End measurement
  const auto end = std::chrono::steady_clock::now();

  // Approximate simulation wall time
  const float elapsedTime = std::chrono::duration<float>(end - start).count();
  std::printf("Time: %f s\n", elapsedTime);


  /********************************************************************************************************************/
  /*                                     TODO: Memory transfer GPU -> CPU                                             */
  /********************************************************************************************************************/

  CUDA_CALL(hipMemcpy(hParticles.posX,   dParticles.posX,   N * sizeof(float), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(hParticles.posY,   dParticles.posY,   N * sizeof(float), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(hParticles.posZ,   dParticles.posZ,   N * sizeof(float), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(hParticles.velX,   dParticles.velX,   N * sizeof(float), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(hParticles.velY,   dParticles.velY,   N * sizeof(float), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(hParticles.velZ,   dParticles.velZ,   N * sizeof(float), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(hParticles.weight, dParticles.weight, N * sizeof(float), hipMemcpyDeviceToHost));


  // Compute reference center of mass on CPU
  const float4 refCenterOfMass = centerOfMassRef(md);

  std::printf("Reference center of mass: %f, %f, %f, %f\n",
              refCenterOfMass.x,
              refCenterOfMass.y,
              refCenterOfMass.z,
              refCenterOfMass.w);

  centerOfMass <<< 32, 64 >>> (dParticles, dFinalCom, dLock, N);
  
  CUDA_CALL(hipMemcpy(&hFinalCom, dFinalCom, sizeof(float4), hipMemcpyDeviceToHost));

  std::printf("Center of mass on GPU: %f, %f, %f, %f\n",
              hFinalCom.x,
              hFinalCom.y,
              hFinalCom.z,
              hFinalCom.w);

  // Writing final values to the file
  h5Helper.writeComFinal(refCenterOfMass);
  h5Helper.writeParticleDataFinal();

  /********************************************************************************************************************/
  /*                                     TODO: GPU side memory deallocation                                           */
  /********************************************************************************************************************/

  CUDA_CALL(hipFree(dParticles.posX));
  CUDA_CALL(hipFree(dParticles.posY));
  CUDA_CALL(hipFree(dParticles.posZ));
  CUDA_CALL(hipFree(dParticles.velX));
  CUDA_CALL(hipFree(dParticles.velY));
  CUDA_CALL(hipFree(dParticles.velZ));
  CUDA_CALL(hipFree(dParticles.weight));

  CUDA_CALL(hipFree(dTmpVelocities.x));
  CUDA_CALL(hipFree(dTmpVelocities.y));
  CUDA_CALL(hipFree(dTmpVelocities.z));

  /********************************************************************************************************************/
  /*                                           CPU side memory deallocation                                           */
  /********************************************************************************************************************/

  operator delete[](hParticles.posX);
  operator delete[](hParticles.posY);
  operator delete[](hParticles.posZ);
  operator delete[](hParticles.velX);
  operator delete[](hParticles.velY);
  operator delete[](hParticles.velZ);
  operator delete[](hParticles.weight);


}// end of main
//----------------------------------------------------------------------------------------------------------------------
